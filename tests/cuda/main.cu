
#include <hip/hip_runtime.h>
#include <stdio.h>  /* Для функции printf() */
#include <stdlib.h> /* Для маркеров статуса */

void
helloFromCPU(void)
{ /* Эта функция работает на хосте */
    printf("Hello World from CPU!\n");
}

__global__ void
helloFromGPU()
{ /* Это ядро запускается на устройстве */
    printf("Hello World from GPU!\n");
}

int
main(int argc, char** argv)
{
    helloFromCPU();           /* Вызов с хоста */
    helloFromGPU<<<1, 1>>>(); /* Запуск с хоста */
    hipDeviceReset();        /* Уборка на устройстве */
    return (EXIT_SUCCESS);
}